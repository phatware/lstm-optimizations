﻿
#include "hip/hip_runtime.h"
#include ""
#include "cuda_utils.cuh"

#include <stdio.h>

extern "C" int rnn_main(int argc, char* argv[]);

int main(int argc, char* argv[])
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus;
    int count = 0;

    cudaStatus = hipGetDeviceCount(&count);
    if (cudaStatus != hipSuccess || count < 1)
    {
        fprintf(stderr, "hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\r\n");
        return -1;
    }
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\r\n");
        return -1;
    }

	int res = rnn_main(argc, argv);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return -1;
    }
    return res;
}
